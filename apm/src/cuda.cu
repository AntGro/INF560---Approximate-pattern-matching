// CUDA runtime
#include <hip/hip_runtime.h>




int __device__ min3(int a, int b, int c) {
    return ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)));
}


int __device__ levenshtein_cuda(char *s1, char *s2, int len, int *column) {
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++) {
        column[y] = y;
    }
    for (x = 1; x <= len; x++) {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++) {
            olddiag = column[y];
            column[y] = min3(
                    column[y] + 1,
                    column[y - 1] + 1,
                    lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1)
            );
            lastdiag = olddiag;

        }
    }
    return (column[len]);
}

void __global__ matchesKernel(int* d_n_matches, char * d_buf, char * d_pattern, int i, int size_pattern, int offset, int n_bytes, int approx_factor){

    /* Traverse the input data up to the end of the file */
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int distance = 0 ;
    int size ;

    size = size_pattern ;
    int* columns = (int *) malloc((size_pattern + 1) * sizeof(int));
    while (j < n_bytes) {
        if (n_bytes - j < size_pattern ){
            size = n_bytes - j ;
        }

        distance = levenshtein_cuda(d_pattern + offset, &d_buf[j], size, columns ) ;
        if ( distance <= approx_factor) {
            atomicAdd(&d_n_matches[i], 1);
        }

        j += stride;
    }
    free(columns);

}

int __host__ gpu_find_matches (int nb_patterns, char** pattern, char * buf, int n_bytes, int* n_matches, int approx_factor) {
/* Check each pattern one by one */
    int i;
    int* d_n_matches;
    char * d_pattern;
    char* d_buf;
    int* offset = (int *)malloc( nb_patterns * sizeof( int ) ) ;
    int* lens = (int *)malloc( nb_patterns * sizeof( int ) ) ;
    int sum_lens;
    lens[0] = strlen(pattern[0]);
    offset[0] = 0;
    sum_lens = lens[0];
    for (i = 1; i < nb_patterns; i++) {
        offset[i] = offset[i-1] + lens[i-1];
        lens[i] = strlen(pattern[i]);
        sum_lens += lens[i];
    }
    char* concat_patterns = (char*) malloc( sum_lens * sizeof( char ) ) ;
    for (i = 0; i < nb_patterns; i++) {
        strcpy (concat_patterns + offset[i], pattern[i]);
    }

    hipMalloc((void **)&d_n_matches, nb_patterns*sizeof(int));
    hipMalloc((void **)&d_pattern, sum_lens*sizeof(char));
    hipMalloc((void **)&d_buf, n_bytes);
    hipMemcpy(d_pattern, concat_patterns, sum_lens*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_buf, buf, n_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_n_matches, n_matches, nb_patterns*sizeof(int), hipMemcpyHostToDevice);

    int Dg = 4;
    int Db = 256;
    for (i = 0; i < nb_patterns; i++) {
        matchesKernel<<<Dg,Db>>>(d_n_matches, d_buf, d_pattern, i, lens[i], offset[i], n_bytes, approx_factor);
    }

    hipMemcpy(n_matches, d_n_matches, nb_patterns*sizeof(int), hipMemcpyDeviceToHost);

    return 0;
}
